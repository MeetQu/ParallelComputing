#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>

#define N 512

__global__ void calcSum(float *d_arr, float *maxArray) {
    unsigned int t = threadIdx.x;
    unsigned int bid = blockIdx.x;
    
    maxArray[t] = d_arr[t + bid * N];
    for (unsigned int stride = blockDim.x / 2; stride > 0;  stride >>= 1) {
        __syncthreads();
        if (t < stride)
            maxArray[t] = maxArray[t + stride] > maxArray[t] ? maxArray[t + stride] : maxArray[t];
    }
}

int main() {
    float *h_arr, *d_arr, *h_maxArray, *d_maxArray;
    int total = N * N;
    int mem_size = total * sizeof(float);

    h_arr = (float *) malloc(mem_size);
    h_maxArray = (float *) malloc(N * sizeof(float));
    for (int i = 0; i < total; i++) {
        h_arr[i] = 3.0;
    }

    for (int i = 0; i < N; i++) {
        h_maxArray[i] = 0.0;
    }

    hipMalloc((void **) &d_arr, mem_size);
    hipMalloc((void **) &d_maxArray, N * sizeof(float));
    hipMemcpy(d_arr, h_arr, mem_size, hipMemcpyHostToDevice);
    hipMemcpy(d_maxArray, h_maxArray, N * sizeof(float), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(N);
    dim3 blocksPerGrid(16);

    // 记录程序开始运行的时间
    float time;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    calcSum <<< blocksPerGrid, threadsPerBlock >>> (d_arr, d_maxArray);

    hipDeviceSynchronize();
    hipEventRecord(stop, 0);
    hipEventSynchronize(start);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("Time elapsed: %.6f ms\n", time);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipMemcpy(h_maxArray, d_maxArray, N * sizeof(float), hipMemcpyDeviceToHost);

    for (int i = 0; i < N; ++i) {
        printf("The max number of row %d :%.f\n", i,  h_maxArray[i]);
    }
    
    hipFree(d_arr);
    hipFree(d_maxArray);
    free(h_arr);
    free(h_maxArray);
    return 0;
}

