#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>

#define N 512

__global__ void getMaxValueOfRow(float *d_arr, float *maxArray) {
    unsigned int t = threadIdx.x;
    unsigned int bid = blockIdx.x;
    __shared__ float ds_arr[N];
    
    ds_arr[t] = d_arr[t + bid * N];
    for (unsigned int stride = 1; stride < blockDim.x;  stride *= 2) {
        __syncthreads();
        ds_arr[t] = ds_arr[t + stride] > ds_arr[t] ? ds_arr[t + stride] : ds_arr[t];
    }
    maxArray[bid % N] = ds_arr[t];
}

int main() {
    float *h_arr, *d_arr, *h_maxArray, *d_maxArray;
    int total = N * N;
    int mem_size = total * sizeof(float);

    h_arr = (float *) malloc(mem_size);
    h_maxArray = (float *) malloc(N * sizeof(float));
    for (int i = 0; i < total; i++) {
        h_arr[i] = 3.0;
    }

    for (int i = 0; i < N; i++) {
        h_maxArray[i] = 0.0;
    }

    hipMalloc((void **) &d_arr, mem_size);
    hipMalloc((void **) &d_maxArray, N * sizeof(float));
    hipMemcpy(d_arr, h_arr, mem_size, hipMemcpyHostToDevice);
    hipMemcpy(d_maxArray, h_maxArray, N * sizeof(float), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(N);
    dim3 blocksPerGrid(N);

    // 记录程序开始运行的时间
    float time;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    
    getMaxValueOfRow <<< blocksPerGrid, threadsPerBlock >>> (d_arr, d_maxArray);

    hipDeviceSynchronize();
    hipEventRecord(stop, 0);
    hipEventSynchronize(start);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("Time elapsed: %.6f ms\n", time);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipMemcpy(h_maxArray, d_maxArray, N * sizeof(float), hipMemcpyDeviceToHost);

    // for (int i = 0; i < N; ++i) {
    //     printf("The max number of row %d :%.f\n", i,  h_maxArray[i]);
    // }
    
    // 验证结果
    int count = 0;
    for (int i = 0; i < N * N; ++i) {
        if (h_maxArray[i] == 3)
            count++;
    }
    printf("count = %d\n", count);
    
    hipFree(d_arr);
    hipFree(d_maxArray);
    free(h_arr);
    free(h_maxArray);
    return 0;
}