#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <iostream>
#include <limits.h>

#define N 8192


// ������ÿ������ȡƽ��
__global__ void get_vector_squared(float* vector, float* vector_squared) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    vector_squared[idx] = vector[idx] * vector[idx];
}


// ������ƽ����������ۼ����
__global__ void get_vector_sqaured_sum_kernel(float *vector_squared, double *vector_squared_sum) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    for (unsigned int stride = gridDim.x * blockDim.x / 2; stride > 0; stride >>= 1) {
        __syncthreads();
        if (idx < stride)
            vector_squared[idx] = vector_squared[idx + stride] + vector_squared[idx];
    }
    *vector_squared_sum = vector_squared[0];
}

// ���ɾ���
__global__ void create_matrix_kernel(float* vector, float* matrix, float vector_mod) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    for (unsigned int i = 0; i < N; i++) {
        matrix[i + idx * N] = (vector[idx] * vector[i]) / vector_mod;
    }
}

// ��ȡ������ÿһ�е����ֵ����Сֵ�����䱣�浽��Ӧ��������
__global__ void get_max_min_kernel(float* matrix, float* min_array, float* max_array) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    for (unsigned int i = 0; i < N; i++) {
        max_array[idx] = matrix[i + idx * N] > max_array[idx] ? matrix[i + idx * N] : max_array[idx];
        min_array[idx] = matrix[i + idx * N] < min_array[idx] ? matrix[i + idx * N] : min_array[idx];
    }
}

// ��ȡ������������ֵ
__global__ void get_max_value_of_row(float* array, float* max) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    for (unsigned int stride = gridDim.x * blockDim.x / 2; stride > 0; stride >>= 1) {
        __syncthreads();
        if (idx < stride)
            array[idx] = array[idx + stride] > array[idx] ? array[idx + stride] : array[idx];
    }
    *max = array[0];
}

// ��ȡ�����������Сֵ
__global__ void get_min_value_of_row(float* array, float* min) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    for (unsigned int stride = gridDim.x * blockDim.x / 2; stride > 0; stride >>= 1) {
        __syncthreads();
        if (idx < stride)
            array[idx] = array[idx + stride] < array[idx] ? array[idx + stride] : array[idx];
    }
    *min = array[0];
}


void get_max_min(float *vector, float *matrix, float min, float max) {
    double r_square_sum = 0;
    for (int i = 0; i < N; i++) {
        r_square_sum += vector[i] * vector[i];
    }

    double r_mod = sqrtf(r_square_sum);
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            matrix[i * N + j] = (vector[i] * vector[j]) / r_mod;
            if (i == 0 && j == 0) {
                max = matrix[0];
                min = matrix[0];
            } else {
                max = matrix[i * N + j] > max ? matrix[i * N + j] : max;
                min = matrix[i * N + j] < min ? matrix[i * N + j] : min;
            }
        }
    }
    printf("CPU => min = %.6f, max = %.6f\n\n", min, max);
}


void init_Data(char* file_path, float* vector) {
    FILE* file;
    file = fopen(file_path, "r");
    for (int i = 0; !feof(file); i++)
        fscanf(file, "%f", &vector[i]);
    fclose(file);
}

int main() {
    float *host_vector, *host_matrix, *host_min_array, *host_max_array;
    float host_min = 0, host_max = 0;
    double vector_mod = 0, vector_squared_sum = 0;

    host_vector = (float*)malloc(sizeof(float) * N);
    host_matrix = (float*)malloc(sizeof(float) * N * N);
    host_min_array = (float*)malloc(sizeof(float) * N);
    host_max_array = (float*)malloc(sizeof(float) * N);
    

    // ��ȡ����
    init_Data("./testdata6.txt", host_vector);
    

    //========================= CPU start =========================
    // ��¼����ʼ���е�ʱ��
    double startTime, endTime;
    startTime = (double)clock();

    // CPU ����
    get_max_min(host_vector, host_matrix, host_min, host_max);

    // ����������л��ѵ�ʱ��
    endTime = (double)clock();
    printf("Time elapsed on CPU: %.6f ms\n\n", endTime - startTime);
    //========================= CPU end ===========================


    //========================= GPU start =========================
    // ��¼����ʼ���е�ʱ��
    float time;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    // GPU ����
    // ��ʼ������
    float* device_vector, *device_vector_squared;
    float* device_matrix;
    double *device_vector_squared_sum;
    float* device_min_array, * device_max_array;

    hipMalloc((void**)&device_vector, sizeof(float) * N);
    hipMalloc((void**)&device_vector_squared, sizeof(float) * N);
    hipMalloc((void**)&device_matrix, sizeof(float) * N * N);
    hipMalloc((void**)&device_vector_squared_sum, sizeof(double));
    hipMalloc((void**)&device_min_array, sizeof(float) * N);
    hipMalloc((void**)&device_max_array, sizeof(float) * N);
    hipMemcpy(device_vector, host_vector, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(device_vector_squared_sum, &vector_squared_sum, sizeof(double), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(1024);
    dim3 blocksPerGrid(8);

    get_vector_squared <<< blocksPerGrid, threadsPerBlock >>> (device_vector, device_vector_squared);
    hipDeviceSynchronize();

    get_vector_sqaured_sum_kernel <<< blocksPerGrid, threadsPerBlock >>> (device_vector_squared, device_vector_squared_sum);
    hipDeviceSynchronize();
    hipMemcpy(&vector_squared_sum, device_vector_squared_sum, sizeof(double), hipMemcpyDeviceToHost);


    // ��������ģ
    vector_mod = sqrtf(vector_squared_sum);
    //printf("Modulus of vector = %.6f\n", vector_mod);


    // ���ɾ���
    create_matrix_kernel <<< blocksPerGrid, threadsPerBlock >>> (device_vector, device_matrix, vector_mod);
    hipDeviceSynchronize();
    hipMemcpy(host_matrix, device_matrix, sizeof(float) * N * N, hipMemcpyDeviceToHost);


    // ��ȡ������ÿ�е����ֵ����Сֵ���洢��һ��һά������
    for (int i = 0; i < N; i++) {
        host_min_array[i] = host_matrix[0];
        host_max_array[i] = host_matrix[0];
    }
    hipMemcpy(device_min_array, host_min_array, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(device_max_array, host_max_array, sizeof(float) * N, hipMemcpyHostToDevice);

    get_max_min_kernel <<< blocksPerGrid, threadsPerBlock >>> (device_matrix, device_min_array, device_max_array);
    hipDeviceSynchronize();
    hipMemcpy(host_min_array, device_min_array, sizeof(float) * N, hipMemcpyDeviceToHost);
    hipMemcpy(host_max_array, device_max_array, sizeof(float) * N, hipMemcpyDeviceToHost);
    

    // ��һά����ʹ���޷�֧��ɢ�Ĳ��й�Լ�����������Сֵ
    float *device_min, *device_max;
    hipMalloc((void**)&device_min, sizeof(float));
    hipMalloc((void**)&device_max, sizeof(float));
    
    get_max_value_of_row <<< blocksPerGrid, threadsPerBlock >>> (device_max_array, device_max);
    hipMemcpy(&host_max, device_max, sizeof(float), hipMemcpyDeviceToHost);
    get_min_value_of_row <<< blocksPerGrid, threadsPerBlock >>> (device_min_array, device_min);
    hipMemcpy(&host_min, device_min, sizeof(float), hipMemcpyDeviceToHost);
    

    // ������
    printf("GPU => min = %.6f, max = %.6f\n\n", host_min, host_max);


    hipDeviceSynchronize();
    hipEventRecord(stop, 0);
    hipEventSynchronize(start);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("Time elapsed on GPU: %.6f ms\n", time);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipFree(device_vector);
    hipFree(device_vector_squared);
    hipFree(device_matrix);
    hipFree(device_vector_squared_sum);
    hipFree(device_min_array);
    hipFree(device_max_array);
    hipFree(device_min);
    hipFree(device_max);
    //========================= GPU end ===========================
    
    free(host_vector);
    free(host_matrix);
    free(host_min_array);
    free(host_max_array);

    return 0;
}