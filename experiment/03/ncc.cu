#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#define N 1024

float standard_ncc(float* mat_f, float* mat_g, int u, int v, int wx, int wy);
float sum_table_ncc(float* mat_f, float* mat_g, int u, int v, int wx, int wy);
float get_sum_of_products(float* mat_a, float* mat_b, float* st, int u, int v);
__global__ void get_mat_sum(float* mat_a, float* mat_b, float* sum, int u, int v, int wx, int wy);

int main()
{
    int count = N * N;
    int roi1_u = 51;
    int roi1_v = 51;
    //int roi2_u = 81;
    //int roi2_v = 91;
    int roi1_wx = 900;
    int roi1_wy = 900;
    //int roi2_wx = 800;
    //int roi2_wy = 800;

    float* mat_f, * mat_g;
    mat_f = (float*)malloc(sizeof(float) * count);
    mat_g = (float*)malloc(sizeof(float) * count);
    if (mat_f == NULL || mat_g == NULL) return -1;

    for (int i = 0; i < count; i++) {
        mat_f[i] = 3.0;
        mat_g[i] = 2.0;
    }

    //========================= CPU start =========================

    // ��¼����ʼ���е�ʱ��
    double start_time, end_time;
    start_time = (double)clock();

    float ncc = standard_ncc(mat_f, mat_g, roi1_u, roi1_v, roi1_wx, roi1_wy);
    printf("standard_ncc_on_CPU\t-> %.6f\n", ncc);
    end_time = (double)clock();
    printf("time elapsed on CPU\t-> %.6f ms\n\n", end_time - start_time);
    start_time = end_time;

    ncc = sum_table_ncc(mat_f, mat_g, roi1_u, roi1_v, roi1_wx, roi1_wy);
    printf("sum_table_ncc_on_CPU\t-> %.6f\n", ncc);

    // ����������л��ѵ�ʱ��
    end_time = (double)clock();
    printf("time elapsed on CPU\t-> %.6f ms\n\n", end_time - start_time);

    //========================= CPU end ===========================


    //========================= GPU start =========================
    // ͨ����׼�������м��� NCC
    // ��¼����ʼ���е�ʱ��
    float time;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    dim3 threads_per_block(N);
    dim3 number_of_blocks(N);

    float* device_mat_f, * device_mat_g;
    hipMalloc((void**)&device_mat_f, sizeof(float) * count);
    hipMalloc((void**)&device_mat_g, sizeof(float) * count);
    hipMemcpy(device_mat_f, mat_f, sizeof(float) * count, hipMemcpyHostToDevice);
    hipMemcpy(device_mat_g, mat_g, sizeof(float) * count, hipMemcpyHostToDevice);

    float* device_fg_sum, * device_f2_sum, * device_g2_sum;
    float fg_sum = 0, f2_sum = 0, g2_sum = 0;
    hipMalloc((void**)&device_fg_sum, sizeof(float));
    hipMalloc((void**)&device_f2_sum, sizeof(float));
    hipMalloc((void**)&device_g2_sum, sizeof(float));
    hipMemcpy(device_fg_sum, &fg_sum, sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(device_f2_sum, &f2_sum, sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(device_g2_sum, &g2_sum, sizeof(float), hipMemcpyHostToDevice);

    get_mat_sum << < number_of_blocks, threads_per_block >> >
        (device_mat_f, device_mat_g, device_fg_sum, roi1_u, roi1_v, roi1_wx, roi1_wy);
    get_mat_sum << < number_of_blocks, threads_per_block >> >
        (device_mat_f, device_mat_f, device_f2_sum, roi1_u, roi1_v, roi1_wx, roi1_wy);
    get_mat_sum << < number_of_blocks, threads_per_block >> >
        (device_mat_g, device_mat_g, device_g2_sum, roi1_u, roi1_v, roi1_wx, roi1_wy);

    hipDeviceSynchronize();
    hipMemcpy(&fg_sum, device_fg_sum, sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&f2_sum, device_f2_sum, sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&g2_sum, device_g2_sum, sizeof(float), hipMemcpyDeviceToHost);

    ncc = fg_sum / sqrtf(f2_sum * g2_sum);
    printf("standard_ncc_on_GPU\t-> %.6f\n", ncc);

    hipDeviceSynchronize();
    hipEventRecord(stop, 0);
    hipEventSynchronize(start);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("time elapsed on GPU\t-> %.6f ms\n", time);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    //========================= GPU end ===========================

    free(mat_f);
    free(mat_g);
    hipFree(device_mat_f);
    hipFree(device_mat_g);
    hipFree(device_fg_sum);
    hipFree(device_f2_sum);
    hipFree(device_g2_sum);
    return 0;
}


// ��׼�������� NCC��CPU��
float standard_ncc(float* mat_f, float* mat_g, int u, int v, int wx, int wy) {
    float product = 0, f_squared_sum = 0, g_squared_sum = 0;

    for (int i = u; i < u + wx - 1 && i < N; i++) {
        for (int j = v; j < v + wy - 1 && j < N; j++) {
            product += mat_f[i * N + j] * mat_g[i * N + j];
            f_squared_sum += powf(mat_f[i * N + j], 2);
            g_squared_sum += powf(mat_g[i * N + j], 2);
        }
    }
    return product / sqrtf((float)f_squared_sum * (float)g_squared_sum);
}


// �ͱ������� NCC��CPU��
float sum_table_ncc(float* mat_f, float* mat_g, int u, int v, int wx, int wy) {
    float* st_f_squared, * st_g_squared, * st_f_g;
    st_f_squared = (float*)malloc(sizeof(int) * N * N);
    st_g_squared = (float*)malloc(sizeof(int) * N * N);
    st_f_g = (float*)malloc(sizeof(int) * N * N);
    if (st_f_squared == NULL || st_g_squared == NULL || st_f_g == NULL)
        return -1;

    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            st_f_squared[i * N + j] = get_sum_of_products(mat_f, mat_f, st_f_squared, i, j);
            st_g_squared[i * N + j] = get_sum_of_products(mat_g, mat_g, st_g_squared, i, j);
            st_f_g[i * N + j] = get_sum_of_products(mat_f, mat_g, st_f_g, i, j);
        }
    }

    float product = 0.0, f_squared_sum = 0.0, g_squared_sum = 0.0;
    product = st_f_g[(u + wx - 1) * N + v + wy - 1]
        - st_f_g[(u - 1) * N + v + wy - 1]
        - st_f_g[(u + wx - 1) * N + v - 1]
        + st_f_g[(u - 1) * N + v - 1];
    f_squared_sum = st_f_squared[(u + wx - 1) * N + v + wy - 1]
        - st_f_squared[(u - 1) * N + v + wy - 1]
        - st_f_squared[(u + wx - 1) * N + v - 1]
        + st_f_squared[(u - 1) * N + v - 1];
    g_squared_sum = st_g_squared[(u + wx - 1) * N + v + wy - 1]
        - st_g_squared[(u - 1) * N + v + wy - 1]
        - st_g_squared[(u + wx - 1) * N + v - 1]
        + st_g_squared[(u - 1) * N + v - 1];

    free(st_f_squared);
    free(st_g_squared);
    free(st_f_g);
    return product / sqrtf((float)f_squared_sum * (float)g_squared_sum);
}


// ����������ĳ˻���CPU��
float get_sum_of_products(float* mat_a, float* mat_b, float* st, int u, int v) {
    if (u == 0 || v == 0)
        return 0;
    else
        return mat_a[u * N + v] * mat_b[u * N + v] + st[(u - 1) * N + v]
        + st[u * N + v - 1] - st[(u - 1) * N + v - 1];
}


// ����Ԫ����˲��������ͣ��޲��й�Լ��
__global__ void get_mat_sum(float* mat_a, float* mat_b, float* sum, int u, int v, int wx, int wy) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= u * N + v && idx <= wx * N + wy)
        atomicAdd(sum, mat_a[idx] * mat_b[idx]);
}