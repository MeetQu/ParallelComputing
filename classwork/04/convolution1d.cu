#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>

__global__ void convolution_1D_basic_kernel(float* in, float* mask, float* out, int mask_size, int input_size) {

    float val = 0;
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int n_start_point = i - (mask_size / 2);

    if (i < input_size) {
        for (int j = 0; j < mask_size; j++) {
            if ((n_start_point + j) >= 0 && (n_start_point + j < input_size)) {
                val += in[n_start_point + j] * mask[j];
            }
        }
        out[i] = val;
    }
}


int main() {
    int input_size = 7, mask_size = 5;
    float* in, * out, * device_in, * device_mask, * device_out;
    float mask[5] = { 3, 4, 5, 4, 3 };

    in = (float*)malloc(input_size * sizeof(float));
    out = (float*)malloc(input_size * sizeof(float));
    for (int i = 0; i < input_size; i++) {
        in[i] = i + 1;
    }

    hipMalloc((void**)&device_in, input_size * sizeof(float));
    hipMalloc((void**)&device_mask, mask_size * sizeof(float));
    hipMalloc((void**)&device_out, input_size * sizeof(float));
    hipMemcpy(device_in, in, input_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(device_mask, mask, mask_size * sizeof(float), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(input_size);
    dim3 blocksPerGrid(1);

    convolution_1D_basic_kernel << < blocksPerGrid, threadsPerBlock >> > (device_in, device_mask, device_out, mask_size, input_size);
    hipDeviceSynchronize();

    hipMemcpy(out, device_out, input_size * sizeof(float), hipMemcpyDeviceToHost);

    for (int i = 0; i < input_size; ++i) {
        printf("%.f\t", out[i]);
    }
    printf("\n");

    free(in);
    free(out);
    hipFree(device_in);
    hipFree(device_mask);
    hipFree(device_out);
    return 0;
}

