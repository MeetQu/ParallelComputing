#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>

__global__ void convolution_2D_basic_kernel(float* in, float* mask, float* out, int mask_width, int width, int height) {

    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;

    if (row < height && col < width) {
        int curr_row, curr_col;
        float val = 0.0;
        int n_start_row = row - (mask_width / 2);
        int n_start_col = col - (mask_width / 2);
        for (int i = 0; i < mask_width; i++) { // row
            for (int j = 0; j < mask_width; j++) { // col
                curr_row = n_start_row + i;
                curr_col = n_start_col + j;
                if (curr_row >= 0 && curr_row < height && curr_col >= 0 && curr_col < width) {
                    val += in[curr_row * width + curr_col] * mask[mask_width * i + j];
                    __syncthreads();
                }
            }
        }
        out[row * width + col] = val;
    }
}



int main() {
    int mask_width = 3, width = 7, height = 7;
    int size = width * height * sizeof(float);
    float in[49] = { 193, 245, 178, 215, 64,  234, 13,
                     70,  87,  228, 65,  157, 73,  135,
                     174, 149, 245, 208, 121, 193, 199,
                     167, 57,  140, 62,  90,  192, 239,
                     41,  192, 35,  237, 212, 97,  33,
                     30,  65,  38,  89,  149, 145, 145,
                     127, 129, 65,  50,  140, 19,  120 };
    float mask[9] = { 1, 2, 1,
                      2, 3, 2,
                      1, 2, 1 };
    float* out = (float*)malloc(size);

    float* device_in, * device_out, * device_mask;
    hipMalloc((void**)&device_in, size);
    hipMalloc((void**)&device_out, size);
    hipMalloc((void**)&device_mask, mask_width * mask_width * sizeof(float));
    hipMemcpy(device_in, in, size, hipMemcpyHostToDevice);
    hipMemcpy(device_mask, mask, mask_width * mask_width * sizeof(float), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(width, height);
    dim3 blocksPerGrid(1);

    convolution_2D_basic_kernel << < blocksPerGrid, threadsPerBlock >> > (device_in, device_mask, device_out, mask_width, width, height);
    hipDeviceSynchronize();

    hipMemcpy(out, device_out, size, hipMemcpyDeviceToHost);
    for (int i = 0; i < height; ++i) {
        for (int j = 0; j < width; j++) {
            printf("%.f\t", out[i * width + j]);
        }
        printf("\n");
    }
    printf("\n");

    free(out);
    hipFree(device_in);
    hipFree(device_out);
    hipFree(device_mask);
    return 0;
}

